// ===------ LapackUtils_api_test6.cu --------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: LapackUtils_getrf


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  double2* a_z;
  int64_t* ipiv_s;
  hipsolverHandle_t handle;
  size_t device_ws_size_s;
  size_t host_ws_size_s;
  hipsolverDnParams_t params;
  void* device_ws_s;
  void* host_ws_s;
  int *info;

  hipsolverDnXgetrf(handle, params, 2, 2, HIP_R_32F, a_s, 2, ipiv_s, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  return 0;
}